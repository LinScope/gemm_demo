#include "hip/hip_runtime.h"

__global__ void static_shared_memory()
{
  __shared__ float array[1024];
}

__global__ void dynamic_shared_memory()
{
  __shared__ float array[];
}

int launcher()
{
  hipStream_t stream;
  hipStreamCreate(&stream);

  // static
  static_shared_memory<<<256, 256, 0, stream>>>();
  
  // dynamic
  dynamic_shared_memory<<<256, 256, 1024 * sizeof(int), stream>>>();

  hipStreamSynchronize(stream);
  cudaStreamDestory(&stream);
}
